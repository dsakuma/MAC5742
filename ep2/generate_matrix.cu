
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MATRIX_ORDER 3 // ordem das matrizes (quadradas)

double randZeroToOne()
{
    return rand() / (RAND_MAX + 1.);
}

int randMToN(int M, int N)
{
    return M + (rand() / ( RAND_MAX / (N-M) ) ) ;
}

// void write_matrix(int n_rows, int n_cols, char filename[])
// {
//   /* open file */
//   FILE *f = fopen(filename, "w");
//   if (f == NULL)
//   {
//       printf("Error opening file!\n");
//       exit(1);
//   }
//   /* print qty rows and cols */
//   fprintf(f, "%d %d\n", n_rows, n_cols);
//
//   /* write matrix */
//   for(int i=1; i<=n_rows; i++)
//   {
//     for(int j=1; j<=n_cols; j++)
//     {
//       double val = randMToN(0,10);
//       if (val >= 0.1)
//         fprintf(f, "%d %d %.1f\n", i, j, val);
//     }
//   }
//   fclose(f);
// }

// void write_matrix()

void write_matrix_list(int n_matrix, char filename[])
{
  /* open file */
  FILE *f = fopen(filename, "w");
  if (f == NULL)
  {
      printf("Error opening file!\n");
      exit(1);
  }
  /* print num matrizes */
  fprintf(f, "%d\n", n_matrix);
  fprintf(f, "***");
  /* write matrix */
  for(int n=0; n<=n_matrix; n++)
  {
    for(int i=1; i<=MATRIX_ORDER; i++)
    {
        fprintf(f, "%d %d %d\n", randMToN(0,10), randMToN(0,10), randMToN(0,10));
    }
    fprintf(f, "***");
  }
  /* close file */
  fclose(f);
}

int main(int argc, char **argv)
{
   printf("Generating matrix list!\n");
   if(argc<=2) {
      printf("Usage: %s <qtd_matrizes> <caminho_list_matrizes>\n", argv[0]);
      exit(1);
   }
   int n_matrix = atoi(argv[1]);
   char* filename = argv[2];
   write_matrix_list(n_matrix, filename);
   return 0;
}
