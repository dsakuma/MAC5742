#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>



int main(int argc, char *argv[])
{
  int a;
  int *ptx;
  int *pty;
  int *pttmp;

  printf("Pointer Example Program : Print Pointer Address\n");
  a = 10;
  ptx = &a;

  printf("\n[a  ]:Value of A = %d", a);
  printf("\n[*ptx]:Value of A = %d", *ptx);
  printf("\n[&a ]:Address of A = %p", &a);
  printf("\n[ptx ]:Address of A = %p", ptx);
  printf("\n[&ptx]:Address of ptx = %p", &ptx);
  printf("\n[ptx ]:Value of ptx = %p", ptx);
  printf("\n[ptx ]:Value of pty = %p", pty);

  return 0;
}
