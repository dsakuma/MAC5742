#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>



int main(int argc, char *argv[])
{
  int a;
  int b;
  int *ptx;
  int *pty;
  int *pttmp;

  printf("Pointer Example Program : Print Pointer Address\n");
  a = 10;
  b = 11;
  ptx = &a;
  pty = &y;

  pttmp = ptx;
  ptx = pty;
  pty = ptx;


  printf("\n[a  ]:Value of A = %d", a);
  printf("\n[*ptx]:Value of A = %d", *ptx);
  printf("\n[&a ]:Address of A = %p", &a);
  printf("\n[ptx ]:Address of A = %p", ptx);
  printf("\n[&ptx]:Address of ptx = %p", &ptx);
  printf("\n[ptx ]:Value of ptx = %p", ptx);
  printf("\n[ptx ]:Value of pty = %p", pty);

  return 0;
}
