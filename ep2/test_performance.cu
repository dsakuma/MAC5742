#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"
#include "reduction_seq.h"

#define MATRIX_ORDER 3

int main(int argc, char *argv[])
{
  int result;
  const char* filename;
  const char* description;
  int *y_cuda;
  int *y_seq ;
  struct timeval t0, t1;

  /* Teste 1: Redução de 10 mil matrizes */
  description = "Redução de 10 mil matrizes";
  //given
  filename = "data/teste_10k.txt";
  write_matrix_list(10000, filename, MATRIX_ORDER);
  //when
  gettimeofday(&t0, NULL);
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  gettimeofday(&t1, NULL);
  // //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);
  printf("tempo: %ld us\nresultado:\n", time_elapsed(t0, t1));

  return 0;
}
