#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"
#include "reduction_seq.h"

#define MATRIX_ORDER 3

int main(int argc, char *argv[])
{
  const char* filename;
  const char* description;
  int *y_cuda;
  int *y_seq ;
  struct timeval t0, t1, t2, t3;

  /* Teste 1: Redução de 10 mil matrizes */
  description = "Redução de 10 mil matrizes";
  //given
  filename = "data/teste_10k.txt";
  write_matrix_list(10000, filename, MATRIX_ORDER);
  //when
  gettimeofday(&t0, NULL);
  reduction_cuda(filename, MATRIX_ORDER);
  gettimeofday(&t1, NULL);
  gettimeofday(&t2, NULL);
  reduction_seq(filename, MATRIX_ORDER);
  gettimeofday(&t3, NULL);
  // //then
  print_performance_test_result(description, time_elapsed(t0, t1), time_elapsed(t2, t3));

  /* Teste 2: Redução de 10 mil matrizes */
  description = "Redução de 100 mil matrizes";
  //given
  filename = "data/teste_100k.txt";
  write_matrix_list(100000, filename, MATRIX_ORDER);
  //when
  gettimeofday(&t0, NULL);
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  gettimeofday(&t1, NULL);
  gettimeofday(&t2, NULL);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  gettimeofday(&t3, NULL);
  // //then
  print_performance_test_result(description, time_elapsed(t0, t1), time_elapsed(t2, t3));

  /* Teste 3: Redução de 10 mil matrizes */
  description = "Redução de 1M matrizes";
  //given
  filename = "data/teste_1M.txt";
  write_matrix_list(1000000, filename, MATRIX_ORDER);
  //when
  gettimeofday(&t0, NULL);
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  gettimeofday(&t1, NULL);
  gettimeofday(&t2, NULL);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  gettimeofday(&t3, NULL);
  // //then
  print_performance_test_result(description, time_elapsed(t0, t1), time_elapsed(t2, t3));

  /* Teste 4: Redução de 10 mil matrizes */
  description = "Redução de 10M matrizes";
  //given
  filename = "data/teste_10M.txt";
  write_matrix_list(10000000, filename, MATRIX_ORDER);
  //when
  gettimeofday(&t0, NULL);
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  gettimeofday(&t1, NULL);
  gettimeofday(&t2, NULL);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  gettimeofday(&t3, NULL);
  // //then
  print_performance_test_result(description, time_elapsed(t0, t1), time_elapsed(t2, t3));

  return 0;
}
