#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"
#include "reduction_seq.h"

#define MATRIX_ORDER 3

int main(int argc, char *argv[])
{
  int result;
  const char* filename;
  const char* description;
  int *y_cuda;
  int *y_seq ;

  /* Test 1: Quantidade ímpar de matrizes */
  description = "Redução de 10 mil matrizes";
  //given
  filename = "data/teste_10k.txt";
  write_matrix_list(10000, filename, MATRIX_ORDER);

  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  // //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  return 0;
}
