#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"
#include "reduction_seq.h"

#define MATRIX_ORDER 3

int main(int argc, char *argv[])
{
  const char* filename;
  const char* description;
  struct timeval t0, t1, t2;

  description = "Redução de 1024 matrizes";
  //given
  filename = "data/teste_1k.txt";
  write_matrix_list(1024, filename, MATRIX_ORDER);
  //when
  gettimeofday(&t0, NULL);
  reduction_cuda(filename, MATRIX_ORDER);
  gettimeofday(&t1, NULL);
  reduction_seq(filename, MATRIX_ORDER);
  gettimeofday(&t2, NULL);
  //then
  print_performance_test_result(description, time_elapsed(t0, t1), time_elapsed(t1, t2));

  /* Teste 1: Redução de 2048 matrizes */
  description = "Redução de 2048 matrizes";
  //given
  filename = "data/teste_2k.txt";
  write_matrix_list(2048, filename, MATRIX_ORDER);
  //when
  gettimeofday(&t0, NULL);
  reduction_cuda(filename, MATRIX_ORDER);
  gettimeofday(&t1, NULL);
  reduction_seq(filename, MATRIX_ORDER);
  gettimeofday(&t2, NULL);
  //then
  print_performance_test_result(description, time_elapsed(t0, t1), time_elapsed(t1, t2));

  /* Teste 2: Redução de 8192 matrizes */
  description = "Redução de 8192 matrizes";
  //given
  filename = "data/teste_8k.txt";
  write_matrix_list(8192, filename, MATRIX_ORDER);
  //when
  gettimeofday(&t0, NULL);
  reduction_cuda(filename, MATRIX_ORDER);
  gettimeofday(&t1, NULL);
  reduction_seq(filename, MATRIX_ORDER);
  gettimeofday(&t2, NULL);
  //then
  print_performance_test_result(description, time_elapsed(t0, t1), time_elapsed(t1, t2));

  /* Teste 3: Redução de 1M matrizes */
  description = "Redução de 1M matrizes";
  //given
  filename = "data/teste_1M.txt";
  write_matrix_list(1000000, filename, MATRIX_ORDER);
  //when
  gettimeofday(&t0, NULL);
  reduction_cuda(filename, MATRIX_ORDER);
  gettimeofday(&t1, NULL);
  reduction_seq(filename, MATRIX_ORDER);
  gettimeofday(&t2, NULL);
  //then
  print_performance_test_result(description, time_elapsed(t0, t1), time_elapsed(t1, t2));

  /* Teste 4: Redução de 10M matrizes */
  description = "Redução de 10M matrizes";
  //given
  filename = "data/teste_10M.txt";
  write_matrix_list(10000000, filename, MATRIX_ORDER);
  //when
  gettimeofday(&t0, NULL);
  reduction_cuda(filename, MATRIX_ORDER);
  gettimeofday(&t1, NULL);
  reduction_seq(filename, MATRIX_ORDER);
  gettimeofday(&t2, NULL);
  //then
  print_performance_test_result(description, time_elapsed(t0, t1), time_elapsed(t1, t2));

  return 0;
}
