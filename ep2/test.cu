#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"
#include "reduction_seq.h"

#define MATRIX_ORDER 3

int main(int argc, char *argv[])
{
  int result;
  const char* filename;
  const char* description;
  int *y_cuda;
  int *y_seq ;

  /* Teste 1: Uma matriz */
  description = "Uma matriz";
  //given
  filename = "data/teste_1.txt";
  write_matrix_list(1, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 2: Duas matrizes */
  description = "Duas matrizes";
  //given
  filename = "data/teste_2.txt";
  write_matrix_list(2, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 3: Quantidade ímpar de matrizes */
  description = "Quantidade ímpar de matrizes";
  //given
  filename = "data/teste_impar.txt";
  write_matrix_list(3, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);


  /* Teste 4: Quantidade par de matrizes */
  description = "Quantidade par de matrizes";
  //given
  filename = "data/teste_par.txt";
  write_matrix_list(4, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 5: 10 matrizes */
  description = "Dez matrizes";
  //given
  filename = "data/teste_10.txt";
  write_matrix_list(10, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 6: 255 matrizes */
  description = "255 matrizes (n de threads por bloco - 1)";
  //given
  filename = "data/teste_255.txt";
  write_matrix_list(255, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 7: 256 matrizes */
  description = "256 matrizes (n de threads por bloco)";
  //given
  filename = "data/teste_256.txt";
  write_matrix_list(256, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 8: 257 matrizes */
  description = "257 matrizes (n de threads por bloco + 1)";
  //given
  filename = "data/teste_257.txt";
  write_matrix_list(257, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 9: 10k matrizes */
  description = "10k matrizes (usa bastante memória da gpu)";
  //given
  filename = "data/teste_10k.txt";
  write_matrix_list(10000, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 9: 100k matrizes */
  description = "100k matrizes (usa bastante memória da gpu)";
  //given
  filename = "data/teste_100k.txt";
  write_matrix_list(100000, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 11: 1M matrizes */
  description = "1M matrizes (usa bastante memória da gpu)";
  //given
  filename = "data/teste_1M.txt";
  write_matrix_list(1000000, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 12: 10M matrizes */
  description = "10M matrizes (usa bastante memória da gpu)";
  //given
  filename = "data/teste_10M.txt";
  write_matrix_list(10000000, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);
  return 0;
}
