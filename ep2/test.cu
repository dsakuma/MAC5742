#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"
#include "reduction_seq.h"

#define MATRIX_ORDER 3

int main(int argc, char *argv[])
{
  int result;
  const char* filename;
  const char* description;
  int *y_cuda;
  int *y_seq ;

  /* Teste 1: Uma matriz */
  description = "Uma matriz";
  //given
  filename = "data/teste_1.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 2: Duas matrizes */
  description = "Duas matrizes";
  //given
  filename = "data/teste_2.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 3: Quantidade ímpar de matrizes */
  description = "Quantidade ímpar de matrizes";
  //given
  filename = "data/teste_impar.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 4: Quantidade par de matrizes */
  description = "Quantidade par de matrizes";
  //given
  filename = "data/teste_par.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 5: 10 matrizes */
  description = "Dez matrizes (a última thread não tem com o que comparar)";
  //given
  filename = "data/teste_10.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 6: 65 matrizes */
  description = "65 matrizes (acontece bus error)";
  //given
  filename = "data/teste_65.txt";
  write_matrix_list(65, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 7: 1024 matrizes */
  description = "1024 matrizes (n max de threads por bloco)";
  //given
  filename = "data/teste_1024.txt";
  write_matrix_list(1024, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 8: 1025 matrizes */
  description = "1025 matrizes (ultrapassa n max de threads por bloco)";
  //given
  filename = "data/teste_1025.txt";
  write_matrix_list(1025, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 9: 10k matrizes */
  description = "10k matrizes (usa bastante memória da gpu)";
  //given
  filename = "data/teste_10k.txt";
  write_matrix_list(10000, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  return 0;
}
