#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"
#include "reduction_seq.h"

#define MATRIX_ORDER 3

int main(int argc, char *argv[])
{
  int result;
  const char* filename;
  const char* description;
  int *y_cuda;
  int *y_seq ;

  /* Teste 1: Uma matriz */
  description = "Uma matriz";
  //given
  filename = "data/teste_1.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 2: Duas matrizes */
  description = "Duas matrizes";
  //given
  filename = "data/teste_2.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 3: Quantidade ímpar de matrizes */
  description = "Quantidade ímpar de matrizes";
  //given
  filename = "data/teste_impar.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 4: Quantidade par de matrizes */
  description = "Quantidade par de matrizes";
  //given
  filename = "data/teste_par.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 5: 10 matrizes */
  description = "Dez matrizes (a última thread não tem com o que comparar)";
  //given
  filename = "data/teste_10.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Teste 6: 10k matrizes */
  description = "Dez matrizes (a última thread não tem com o que comparar)";
  //given
  filename = "data/teste_10k.txt";
  write_matrix_list(10000, filename, MATRIX_ORDER);
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  return 0;
}
