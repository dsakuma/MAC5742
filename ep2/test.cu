#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"
#include "reduction_seq.h"

#define MATRIX_ORDER 3

int assert_vector(int* a, int* b, int size)
{
  for(int i=0; i<size; i++){
    if(a[i] != b[i])
      return 1;
  }
  return 0;
}

void print_test_result(int test_number, int result)
{
    if(result == 1){
      printf("Test %d failed!\n", test_number);
      return;
    }
    printf("Test %d passed!\n", test_number);
    return;
}


int main(int argc, char *argv[])
{
  int result;

  /* Test 1: Quantidade ímpar de matrizes */
  //given
  char* filename = "data/teste_impar.txt";
  //when
  int *y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  int *y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(1, result);



  return 0;
}
