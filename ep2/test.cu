#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"
#include "reduction_seq.h"

#define MATRIX_ORDER 3

int main(int argc, char *argv[])
{
  int result;
  const char* filename;
  const char* description;
  int *y_cuda;
  int *y_seq ;

  /* Test 1: Quantidade ímpar de matrizes */
  description = "Quantidade ímpar de matrizes";
  //given
  filename = "data/teste_impar.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Test 2: Quantidade par de matrizes */
  description = "Quantidade par de matrizes";
  //given
  filename = "data/teste_par.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  return 0;
}
