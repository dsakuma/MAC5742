#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"
#include "reduction_seq.h"

#define MATRIX_ORDER 3

int assert_vector(int* a, int* b, int size)
{
  for(int i=0; i<size; i++){
    if(a[i] != b[i])
      return 1;
  }
  return 0;
}

void print_test_result(const char description[], int result)
{
    if(result == 1){
      printf("Teste: %s [Falhou]\n", description);
      return;
    }
    printf("Teste: %s [OK]\n", description);
    return;
}

int main(int argc, char *argv[])
{
  int result;
  const char* filename;
  const char* description;
  int *y_cuda;
  int *y_seq ;

  /* Test 1: Quantidade ímpar de matrizes */
  description = "Quantidade ímpar de matrizes";
  //given
  filename = "data/teste_impar.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  /* Test 2: Quantidade par de matrizes */
  description = "Quantidade par de matrizes";
  //given
  filename = "data/teste_par.txt";
  //when
  y_cuda = reduction_cuda(filename, MATRIX_ORDER);
  y_seq = reduction_seq(filename, MATRIX_ORDER);
  //then
  result = assert_vector(y_cuda, y_seq, MATRIX_ORDER*MATRIX_ORDER);
  print_test_result(description, result);

  return 0;
}
