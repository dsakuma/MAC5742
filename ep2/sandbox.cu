#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction.h"

#define D 3 // ordem das matrizes (quadradas)

int main(int argc, char *argv[])
{
    char* filename = argv[1];
    int n_els = D*D;

    int *result = cudaReduction(filename, D);

    print_vector(result, n_els, D);

    return 0;
}
