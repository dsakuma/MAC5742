/**

Ana Martinazzo (7209231)
Daniel

EP2 - redução em CUDA

**/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define BLOCK_SIZE 64   // tamanho do bloco
#define D 3             // dimensão das matrizes (quadradas)

void print_matrix(int** matrix, int n_rows, int n_cols);
void print_vector(int* vector, int n_els);

__global__
void add(int **x, int *y, int n_els, int n_mat)
{
  for (int i = 0; i < n_els; i++)
    y[i] = x[i][0];
}

int main(int argc, char *argv[])
{
    printf("HELLO!!!!");
    int **x;
    int *y;
    int n_els = 9;
    int n_mat;

    FILE *fp;
    int val1, val2, val3;

    // (*input)[i] = (int *) calloc(*n_els, sizeof(int));
    hipError_t err = hipMallocManaged(&x, n_els * sizeof(int));
    err = hipMallocManaged(&y, n_els * sizeof(int));

    fp = fopen("teste.txt", "r");
    fscanf(fp, "%d", &n_mat);
    for(int i=0; i < n_els; i++){
      err = hipMallocManaged(&x[i], n_mat * sizeof(int));
    }

    fscanf(fp, "%*s", NULL); // pula linha

    for(int i=0; i < n_mat; i++)
    {
      printf("i: %d\n", i);
      for(int j=0; j < D; j++)
        {
            printf("j: %d\n", j);

            fscanf(fp, "%d %d %d", &val1, &val2, &val3);
            x[D*j][i] = val1;
            x[D*j+1][i] = val2;
            x[D*j+2][i] = val3;
        }
        fscanf(fp, "%*s", NULL); // pula linha
    }

    print_matrix(x, n_els, n_mat);
    add<<<1,1>>>(x, y, n_els, n_mat);
    hipDeviceSynchronize();
    printf("y:\n");
    print_vector(y, n_els);

    return 0;

}


void print_matrix(int** matrix, int n_rows, int n_cols)
{
  printf("Printing matrix...\n");
  for(int i=0; i<n_rows; i++){
    for(int j=0; j<n_cols; j++){
        printf("%d ", matrix[i][j]);
      }
      printf("\n");
  }
}

void print_vector(int* vector, int n_els)
{
  printf("Printing vector...\n");
  for(int i=0; i<n_els; i++){
    printf("%d ", vector[i]);
    printf("\n");
  }
}
