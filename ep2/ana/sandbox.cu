/**

Ana Martinazzo (7209231)
Daniel

EP2 - redução em CUDA

**/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define BLOCK_SIZE 64   // tamanho do bloco
#define D 3             // dimensão das matrizes (quadradas)

void print_matrix(int** matrix, int n_rows, int n_cols);



int main(int argc, char *argv[])
{
    printf("HELLO!!!!");
    int **x;
    int D = 3;
    int n_els = 9;
    int n_mat = 3;

    FILE *fp;
    int val1, val2, val3;

    // (*input)[i] = (int *) calloc(*n_els, sizeof(int));
    hipError_t err = hipMallocManaged(&x, n_els * sizeof(int));

    for(int i=0; i < n_els; i++){
      hipError_t err = hipMallocManaged(&x[i], n_mat * sizeof(int));
    }

    // fp = fopen("teste.txt", "r");
    // fscanf(fp, "%d", n_mat);
    // fscanf(fp, "%*s", NULL); // pula linha
    //
    // for(int i=0; i < n_mat; i++)
    // {
    //   for(int j=0; j < D; j++)
    //     {
    //         fscanf(fp, "%d %d %d", &val1, &val2, &val3);
    //         x[D*j][i] = val1;
    //         x[D*j+1][i] = val2;
    //         x[D*j+2][i] = val3;
    //     }
    //     fscanf(fp, "%*s", NULL); // pula linha
    // }
    //
    // print_matrix(x, n_els, n_mat);

    return 0;

}


void print_matrix(int** matrix, int n_rows, int n_cols)
{
  printf("Printing matrix...\n");
  for(int i=0; i<n_rows; i++){
    for(int j=0; j<n_cols; j++){
        printf("%d ", matrix[i][j]);
      }
      printf("\n");
  }
}
