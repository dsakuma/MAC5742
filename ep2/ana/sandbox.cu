/**

Ana Martinazzo (7209231)
Daniel

EP2 - redução em CUDA

**/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define BLOCK_SIZE 64   // tamanho do bloco
#define D 3             // dimensão das matrizes (quadradas)

void print_matrix(int** matrix, int n_rows, int n_cols);



int main(int argc, char *argv[])
{

    int *x;
    int n_els = 2;

    // (*input)[i] = (int *) calloc(*n_els, sizeof(int));
    hipError_t err = hipMallocManaged(&x, n_els * sizeof(int));

    return 0;

}


void print_matrix(int** matrix, int n_rows, int n_cols)
{
  printf("Printing matrix...\n");
  for(int i=0; i<n_rows; i++){
    for(int j=0; j<n_cols; j++){
        printf("%d ", matrix[i][j]);
      }
      printf("\n");
  }
}