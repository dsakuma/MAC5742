/**

Ana Martinazzo (7209231)
Daniel

EP2 - redução em CUDA

**/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define BLOCK_SIZE 64   // tamanho do bloco
#define D 3             // dimensão das matrizes (quadradas)

void print_matrix(int** matrix, int n_rows, int n_cols);



int main(int argc, char *argv[])
{
    printf("HELLO!!!!");
    int **x;
    int n_els = 9;
    int n_mat = 3;

    // (*input)[i] = (int *) calloc(*n_els, sizeof(int));
    hipError_t err = hipMallocManaged(&x, n_els * sizeof(int));

    for(int i=0; i < n_els; i++){
      hipError_t err = hipMallocManaged(&x[i], n_mat * sizeof(int));
    }

    // print_matrix(x, 1, 2);

    return 0;

}


void print_matrix(int** matrix, int n_rows, int n_cols)
{
  printf("Printing matrix...\n");
  for(int i=0; i<n_rows; i++){
    for(int j=0; j<n_cols; j++){
        printf("%d ", matrix[i][j]);
      }
      printf("\n");
  }
}
