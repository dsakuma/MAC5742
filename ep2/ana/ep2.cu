#include "hip/hip_runtime.h"
/**

Ana Martinazzo (7209231)
Daniel

EP2 - redução em CUDA

**/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define BLOCK_SIZE 64   // tamanho do bloco
#define D 3             // dimensão das matrizes (quadradas)

void read_file(char *filename, int ***input, int *n_els);
long time_elapsed (struct timeval t0, struct timeval t1);
void print_matrix(int** matrix, int n_rows, int n_cols);

__global__ void min_reduction(int *input, int *output, int pos)
{
    int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    int fst, snd;

    while (number_of_threads > 0)
    {
        if (tid < number_of_threads)
        {
            fst = tid * step_size * 2;
            snd = fst + step_size;
            input[fst] = input[fst] < input[snd] ? input[fst] : input[snd];
        }

        step_size <<= 1;
        number_of_threads >>= 1;
    }

    if(tid == 0)
    {
        output[pos] = input[0];
    }
}


int main(int argc, char *argv[])
{

    if(argc != 2)
    {
        printf("parâmetros requeridos: main <<caminho_matrizes>\n");
        return 1;
    }

    int **host;
    int *result;
    int **dev;
    int *dev_result;
    int n_els;          // quantidade de matrizes
    int i;
    struct timeval t0, t1;

    host = (int **) calloc(D*D, sizeof(int *));
    result = (int *) calloc(D*D, sizeof(int));

    hipMallocManaged(&host, D*D*sizeof(int *));
    hipMallocManaged(&result, D*D*sizeof(int));

    // read_file(argv[1], &host, &n_els);
    // print_matrix(host, D*D, n_els);

    gettimeofday(&t0, NULL);

    // Allocate Unified Memory – accessible from CPU or GPU
    // hipMallocManaged(&x, N*sizeof(float));
    // hipMallocManaged(&y, N*sizeof(float));

    // Run kernel on 1M elements on the GPU
    // add<<<1, 1>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // aloca memoria na gpu
    // hipMalloc((void**) &dev, D*D * sizeof(int *));
    // hipMalloc((void**) &dev_result, D*D * sizeof(int));
    // for(i=0; i < D*D; i++)
    // {
    //     hipMalloc((void**) &dev[i], n_els * sizeof(int));
    // }

    printf("oi 1\n");

    // copiar array 2d pra gpu: http://www.orangeowlsolutions.com/archives/613
    // copia entrada da cpu para a gpu
    // hipMemcpy2D(devPtr, devPitch, hostPtr, hostPitch, Ncols * sizeof(float), Nrows, hipMemcpyHostToDevice)
    // hipMemcpy(dev, host, D*D * n_els * sizeof(int), hipMemcpyHostToDevice);

    printf("oi 2\n");

    // executa o kernel
    // for(i=0; i<D*D; i++)
    // {
    //     min_reduction<<<1, n_els/2>>>(dev[i], dev_result, i);
    // }

    // copia resultado da gpu para a cpu
    // hipMemcpy(result, dev_result, D*D * sizeof(int), hipMemcpyDeviceToHost);

    // limpa memoria
    hipFree(host);
    hipFree(result);
    // hipFree(dev);
    // hipFree(dev_result);

    hipDeviceReset();

    gettimeofday(&t1, NULL);

    printf("tempo: %ld us\nresultado:\n", time_elapsed(t0, t1));
    for(int i=0; i < D; i++)
        printf("%d %d %d\n", result[D*i], result[D*i+1], result[D*i+2]);

    return 0;

}


void read_file(char *filename, int ***input, int *n_els)
{
    FILE *fp;
    int val1, val2, val3;
    int i, j;

    fp = fopen(filename, "r");
    fscanf(fp, "%d", n_els);
    fscanf(fp, "%*s", NULL); // pula linha

    // for(i=0; i < D*D; i++)
    // {
    //     // (*input)[i] = (int *) calloc(*n_els, sizeof(int));
    //     // hipMallocManaged((*input)[i], *n_nels * sizeof(int));
    // }

    for(j=0; j < *n_els; j++)
    {
      for(i=0; i < D; i++)
        {
            fscanf(fp, "%d %d %d", &val1, &val2, &val3);
            (*input)[D*i][j] = val1;
            (*input)[D*i+1][j] = val2;
            (*input)[D*i+2][j] = val3;
        }
        fscanf(fp, "%*s", NULL); // pula linha
    }

    fclose(fp);
}


long time_elapsed (struct timeval t0, struct timeval t1)
{
    return (t1.tv_sec-t0.tv_sec)*1000000 + t1.tv_usec-t0.tv_usec;
}

void print_matrix(int** matrix, int n_rows, int n_cols)
{
  printf("Printing matrix...\n");
  for(int i=0; i<n_rows; i++){
    for(int j=0; j<n_cols; j++){
        printf("%d ", matrix[i][j]);
      }
      printf("\n");
  }
}
