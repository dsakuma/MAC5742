#include "hip/hip_runtime.h"
/**

Ana Martinazzo (7209231)
Daniel

EP2 - redução em CUDA

**/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define BLOCK_SIZE 64   // tamanho do bloco
#define D 3             // dimensão das matrizes (quadradas)

void read_file(char *filename, int ***input, int *n_els);
long time_elapsed (struct timeval t0, struct timeval t1);
void print_matrix(int** matrix, int n_rows, int n_cols);

__global__ void min_reduction(int *input, int *output, int pos)
{
    int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    int fst, snd;

    while (number_of_threads > 0)
    {
        if (tid < number_of_threads)
        {
            fst = tid * step_size * 2;
            snd = fst + step_size;
            input[fst] = input[fst] < input[snd] ? input[fst] : input[snd];
        }

        step_size <<= 1;
        number_of_threads >>= 1;
    }

    if(tid == 0)
    {
        output[pos] = input[0];
    }
}


int main(int argc, char *argv[])
{

    if(argc != 2)
    {
        printf("parâmetros requeridos: main <<caminho_matrizes>\n");
        return 1;
    }

    int **host;
    int *result;
    int **dev;
    int *dev_result;
    int n_els;          // quantidade de matrizes
    int i;
    struct timeval t0, t1;

    host = (int **) calloc(D*D, sizeof(int *));
    result = (int *) calloc(D*D, sizeof(int));
    read_file(argv[1], &host, &n_els);
    print_matrix(host, D*D, n_els);

    gettimeofday(&t0, NULL);

    // aloca memoria na gpu
    hipMalloc((void**) &dev, D*D * sizeof(int *));
    hipMalloc((void**) &dev_result, D*D * sizeof(int));
    for(i=0; i < D*D; i++)
    {
        hipMalloc((void**) &dev[i], n_els * sizeof(int));
    }

    // copiar array 2d pra gpu: http://www.orangeowlsolutions.com/archives/613

    // copia entrada da cpu para a gpu
    hipMemcpy(dev, host, D*D * n_els * sizeof(int), hipMemcpyHostToDevice);

    // executa o kernel
    for(i=0; i<D*D; i++)
    {
        min_reduction<<<1, n_els/2>>>(dev[i], dev_result, i);
    }

    // copia resultado da gpu para a cpu
    hipMemcpy(result, dev_result, D*D * sizeof(int), hipMemcpyDeviceToHost);

    // limpa memoria
    free(host);
    free(result);
    hipFree(dev);
    hipFree(dev_result);

    hipDeviceReset();

    gettimeofday(&t1, NULL);

    printf("tempo: %ld us\nresultado:\n", time_elapsed(t0, t1));
    for(int i=0; i < D; i++)
        printf("%d %d %d\n", result[D*i], result[D*i+1], result[D*i+2]);

    return 0;

}


void read_file(char *filename, int ***input, int *n_els)
{
    FILE *fp;
    int val1, val2, val3;
    int i, j;

    fp = fopen(filename, "r");
    fscanf(fp, "%d", n_els);
    fscanf(fp, "%*s", NULL); // pula linha

    for(i=0; i < D*D; i++)
    {
        (*input)[i] = (int *) calloc(*n_els, sizeof(int));
    }

    for(j=0; j < *n_els; j++)
    {
      for(i=0; i < D; i++)
        {
            fscanf(fp, "%d %d %d", &val1, &val2, &val3);
            (*input)[D*i][j] = val1;
            (*input)[D*i+1][j] = val2;
            (*input)[D*i+2][j] = val3;
        }
        fscanf(fp, "%*s", NULL); // pula linha
    }

    fclose(fp);
}


long time_elapsed (struct timeval t0, struct timeval t1)
{
    return (t1.tv_sec-t0.tv_sec)*1000000 + t1.tv_usec-t0.tv_usec;
}

void print_matrix(int** matrix, int n_rows, int n_cols)
{
  printf("Printing matrix...\n");
  for(int i=0; i<n_rows; i++){
    for(int j=0; j<n_cols; j++){
        printf("%d ", matrix[i][j]);
      }
      printf("\n");
  }
}
