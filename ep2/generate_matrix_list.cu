
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MATRIX_ORDER 3 // ordem das matrizes (quadradas)

double randZeroToOne()
{
    return rand() / (RAND_MAX + 1.);
}

int randMToN(int M, int N)
{
    return M + (rand() / ( RAND_MAX / (N-M) ) ) ;
}


void write_matrix_list(int n_matrix, char filename[])
{
  /* open file */
  FILE *f = fopen(filename, "w");
  if (f == NULL)
  {
      printf("Error opening file!\n");
      exit(1);
  }
  /* print num matrizes */
  fprintf(f, "%d\n", n_matrix);
  fprintf(f, "***\n");
  /* write matrix */
  for(int n=1; n<=n_matrix; n++)
  {
    for(int i=1; i<=MATRIX_ORDER; i++)
    {
        fprintf(f, "%d %d %d\n", randMToN(0,10), randMToN(0,10), randMToN(0,10));
    }
    fprintf(f, "***\n");
  }
  /* close file */
  fclose(f);
}

int main(int argc, char **argv)
{
   printf("Generating matrix list!\n");
   if(argc<=2) {
      printf("Usage: %s <qtd_matrizes> <caminho_list_matrizes>\n", argv[0]);
      exit(1);
   }
   int n_matrix = atoi(argv[1]);
   char* filename = argv[2];
   write_matrix_list(n_matrix, filename);
   return 0;
}
