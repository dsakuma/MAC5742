/**
Ana Martinazzo (7209231)
Daniel Sakuma (5619562)
EP2 - Redução em CUDA
**/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction.h"
#include "reduction_seq.h"

#define D 3 // ordem das matrizes (quadradas)

int main(int argc, char *argv[])
{
    char* filename = argv[1];
    int n_els = D*D;

    int *result = cudaReduction(filename, D);
    int *resultSeq = seqReduction(filename, D);

    print_vector(result, n_els, D);
    print_vector(resultSeq, n_els, D);

    return 0;
}
