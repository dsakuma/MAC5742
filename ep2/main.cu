/**
Ana Martinazzo (7209231)
Daniel
EP2 - redução em CUDA
**/

// ssh -R 52698:localhost:52698 anamartinazzo@shell.vision.ime.usp.br
// https://github.com/AJcodes/cuda_minmax/blob/master/cuda_minmax/kernel.cu

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define CUDA_SAFE_CALL(err) __cuda_safe_call(err, __FILE__, __LINE__)

#define D 3             // dimensão das matrizes (quadradas)
// #define inf 0x7f800000

int get_num_threads(int n);
void print_matrix(int** matrix, int n_rows, int n_cols);
void print_vector(int* vector, int n_els);

inline void
__cuda_safe_call (hipError_t err, const char *filename, const int line_number)
{
  if (err != hipSuccess)
    {
      printf ("CUDA error %i at %s:%i: %s\n",
          err, filename, line_number, hipGetErrorString (err));
      exit (-1);
    }
}


__global__ void min_kernel(int *result, int **input, int n_mat)
{
  //input (9x4)
  //tid=0 (prineira thread)
  //index=0 (primeiro elemente cada matriz)
  //input[index] = [4 1 2 2]
  //mintile[tid] = 4
  //
	__shared__ int mintile[1000];
	unsigned int tid = threadIdx.x;
	unsigned int index = blockIdx.x;
	mintile[tid] = input[index][2*tid];
  if(mintile[tid] > 0)
	  printf("i=%d, tid=%d, part_min=%d\n", index, tid, mintile[tid]);
	__syncthreads();

	// strided index and non-divergent branch
	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
		int index = 2*s*tid;
		if (index < blockDim.x)
		{
			if (mintile[tid + s] < mintile[tid])
				mintile[tid] = mintile[tid + s];
		}
		__syncthreads();
	}
  if(mintile[tid] > 0)
    printf("i=%d, tid=%d, part_min=%d\n", index, tid, mintile[tid]);

	if (tid == 0)
	{
		result[index] = mintile[0];
	}
}


int main(int argc, char *argv[])
{
    int **x;
    int *y;
    int n_els = D*D;
    int n_mat;

    FILE *fp;
    int val1, val2, val3;

    CUDA_SAFE_CALL(hipMallocManaged(&x, n_els * sizeof(int)));
    CUDA_SAFE_CALL(hipMallocManaged(&y, n_els * sizeof(int)));

    fp = fopen(argv[1], "r");
    fscanf(fp, "%d", &n_mat);
    for(int i=0; i < n_els; i++){
      CUDA_SAFE_CALL(hipMallocManaged(&x[i], n_mat * sizeof(int)));
    }

    fscanf(fp, "%*s"); // pula linha

    for(int i=0; i < n_mat; i++)
    {
      for(int j=0; j < D; j++)
        {
            fscanf(fp, "%d %d %d", &val1, &val2, &val3);
            x[D*j][i] = val1;
            x[D*j+1][i] = val2;
            x[D*j+2][i] = val3;
        }
        fscanf(fp, "%*s"); // pula linha
    }

    print_matrix(x, n_els, n_mat);

    dim3 numBlocks(D*D);
    dim3 threadsPerBlock(ceil(n_mat/2));

    // <<<number_of_blocks, block_size>>>
	min_kernel<<<numBlocks, threadsPerBlock>>>(y, x, n_mat);

    hipDeviceSynchronize();

    printf("y:\n");
    print_vector(y, n_els);

    return 0;

}


int get_num_threads(int n)
{
	// TODO: OTIMIZAR PRA WARP COM 32 THREADS ETC
	return ceil(n/2);
}


void print_matrix(int** matrix, int n_rows, int n_cols)
{
	printf("Printing matrix...\n");
	for(int i=0; i<n_rows; i++)
	{
		for(int j=0; j<n_cols; j++)
		{
			printf("%d ", matrix[i][j]);
		}
		printf("\n");
	}
}

void print_vector(int* vector, int n_els)
{
	printf("Printing vector...\n");
	for(int i=0; i<n_els; i++)
	{
		printf("%d ", vector[i]);
		if((i+1)%D == 0)
			printf("\n");
	}
	printf("\n");
}
