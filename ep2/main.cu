/**
Ana Martinazzo (7209231)
Daniel Sakuma (5619562)
EP2 - Redução em CUDA
**/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"
#include "reduction_seq.h"

#define MATRIX_ORDER 3 // ordem das matrizes (quadradas)

int main(int argc, char *argv[])
{
    const char* filename = argv[1];

    // int *y_cuda = reduction_cuda(filename, MATRIX_ORDER);
    // int *y_seq = reduction_seq(filename, MATRIX_ORDER);

    // print_vector(y_cuda, MATRIX_ORDER);
    // print_vector(y_seq, MATRIX_ORDER);

    return 0;
}
