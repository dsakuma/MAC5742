/**
Ana Martinazzo (7209231)
Daniel Sakuma (5619562  )
EP2 - Redução em CUDA
**/


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define CUDA_SAFE_CALL(err) __cuda_safe_call(err, __FILE__, __LINE__)

#define D 3 // dimensão das matrizes (quadradas)

void print_matrix(int** matrix, int n_rows, int n_cols);
void print_vector(int* vector, int n_els);

inline void
__cuda_safe_call (hipError_t err, const char *filename, const int line_number)
{
  if (err != hipSuccess)
    {
      printf ("CUDA error %i at %s:%i: %s\n",
          err, filename, line_number, hipGetErrorString (err));
      exit (-1);
    }
}


__global__ void min_kernel(int *result, int **input, int n_mat)
{
	__shared__ int mintile[3];
  // for(int i=0; i<n_mat; i++)
  //   mintile[i] = 99;
	unsigned int tid = threadIdx.x;
	unsigned int index = blockIdx.x;
	mintile[tid] = input[index][tid];
  // if(mintile[tid] > 0)
	//   printf("i=%d, tid=%d, part_min=%d\n", index, tid, mintile[tid]);

  __syncthreads();
  if(tid == 0 && index ==0)
  {
    printf("mintile inicial tid=%d index=%d:\n", tid, index);
    for(int i=0; i<n_mat; i++)
      printf("%d ", mintile[i]);
    printf("\n");
  }

	// strided index and non-divergent branch
	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
    int idx = 2*s*tid;
    // if(mintile[tid] > 0)
    //   printf("Dentro for: i=%d, tid=%d, s=%d, blockDim=%d\n",
    //        idx, tid, s, blockDim.x);
		if (idx < blockDim.x-1)
		{
      // if(mintile[tid] > 0)
      //   printf("primeiro if, i=%d, tid=%d, mintile[tid]=%d, input[tid + s]=%d\n", idx, tid, mintile[tid], input[index][2*tid + s]);
			if (mintile[idx + s] < mintile[idx])
      {
        // printf("Dentro if: i=%d, idx=%d, s=%d, mintile[idx]=%d, mintile[idx + s]=%d\n",
        //        idx, idx, s, mintile[idx], input[index][2*idx + s]);
        mintile[idx] = mintile[idx + s];
      }
		}
    // if(mintile[tid] > 0)
    // {
    //   printf("mintile:\n");
    //   for(int i=0; i<n_mat; i++)
    //     printf("%d ", mintile[i]);
    //   printf("\n");
    // }
		__syncthreads();
    if(tid == 0 && index ==0)
    {
      printf("mintile tid=%d index=%d, s=%d\n", tid, index, s);
      for(int i=0; i<n_mat; i++)
        printf("%d ", mintile[i]);
      printf("\n");
    }
	}
  // if(mintile[tid] > 0)
  //   printf("i=%d, tid=%d, part_min=%d\n", index, tid, mintile[tid]);

	if (tid == 0)
	{
		result[index] = mintile[0];
	}
}


int main(int argc, char *argv[])
{
    int **x;
    int *y;
    int n_els = D*D;
    int n_mat;

    FILE *fp;
    int val1, val2, val3;

    CUDA_SAFE_CALL(hipMallocManaged(&x, n_els * sizeof(int)));
    CUDA_SAFE_CALL(hipMallocManaged(&y, n_els * sizeof(int)));

    fp = fopen(argv[1], "r");
    fscanf(fp, "%d", &n_mat);



    for(int i=0; i < n_els; i++){
      CUDA_SAFE_CALL(hipMallocManaged(&x[i], n_mat * sizeof(int)));
    }

    fscanf(fp, "%*s"); // skip line

    for(int i=0; i < n_mat; i++)
    {
      for(int j=0; j < D; j++)
      {
          fscanf(fp, "%d %d %d", &val1, &val2, &val3);
          x[D*j][i] = val1;
          x[D*j+1][i] = val2;
          x[D*j+2][i] = val3;
      }
        fscanf(fp, "%*s");  // skip line
    }

    // print_matrix(x, n_els, n_mat);

    dim3 numBlocks(D*D);
    dim3 threadsPerBlock(n_mat);

	  min_kernel<<<numBlocks, threadsPerBlock>>>(y, x, n_mat); //<<<number_of_blocks, block_size>>>

    hipDeviceSynchronize();

    printf("y:\n");
    print_vector(y, n_els);

    return 0;

}

void print_matrix(int** matrix, int n_rows, int n_cols)
{
	printf("Printing matrix...\n");
	for(int i=0; i<n_rows; i++)
	{
		for(int j=0; j<n_cols; j++)
		{
			printf("%d ", matrix[i][j]);
		}
		printf("\n");
	}
}

void print_vector(int* vector, int n_els)
{
	printf("Printing vector...\n");
	for(int i=0; i<n_els; i++)
	{
		printf("%d ", vector[i]);
		if((i+1)%D == 0)
			printf("\n");
	}
	printf("\n");
}
