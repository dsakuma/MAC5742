/**
Ana Martinazzo (7209231)
Daniel Sakuma (5619562)
EP2 - Redução em CUDA
**/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction.h"

#define D 3 // ordem das matrizes (quadradas)

int main(int argc, char *argv[])
{
    char* filename = argv[1];
    int n_els = D*D;

    int *result = cudaReduction(filename);

    print_vector(result, n_els, D);

    return 0;
}
