#include "hip/hip_runtime.h"
/**
Ana Martinazzo (7209231)
Daniel Sakuma (5619562  )
EP2 - Redução em CUDA
**/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"

#define CUDA_SAFE_CALL(err) __cuda_safe_call(err, __FILE__, __LINE__)

#define D 3 // ordem das matrizes (quadradas)

inline void
__cuda_safe_call (hipError_t err, const char *filename, const int line_number)
{
  if (err != hipSuccess)
    {
      printf ("CUDA error %i at %s:%i: %s\n",
          err, filename, line_number, hipGetErrorString (err));
      exit (-1);
    }
}


__global__ void min_kernel(int *result, int **input, int n_mat)
{
	__shared__ int mintile[3];

	unsigned int tid = threadIdx.x;
	unsigned int index = blockIdx.x;
	mintile[tid] = input[index][tid];

  __syncthreads();
  // if(tid == 0 && index ==0)
  // {
  //   printf("mintile inicial tid=%d index=%d:\n", tid, index);
  //   for(int i=0; i<n_mat; i++)
  //     printf("%d ", mintile[i]);
  //   printf("\n");
  // }

	// strided index and non-divergent branch
	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
    int idx = 2*s*tid;
		if (idx < blockDim.x-1)
		{
			if (mintile[idx + s] < mintile[idx])
      {
        mintile[idx] = mintile[idx + s];
      }
		}
		__syncthreads();
    // if(tid == 0 && index ==0)
    // {
    //   printf("mintile tid=%d index=%d, s=%d\n", tid, index, s);
    //   for(int i=0; i<n_mat; i++)
    //     printf("%d ", mintile[i]);
    //   printf("\n");
    // }
	}

	if (tid == 0)
	{
		result[index] = mintile[0];
	}
}

void cudaReduction(int* result, char filename[])
{
  int **x;
  // int *y;
  int n_els = D*D;
  int n_mat;

  FILE *fp;
  int val1, val2, val3;

  CUDA_SAFE_CALL(hipMallocManaged(&x, n_els * sizeof(int)));
  CUDA_SAFE_CALL(hipMallocManaged(&result, n_els * sizeof(int)));

  fp = fopen(filename, "r");
  fscanf(fp, "%d", &n_mat);

  for(int i=0; i < n_els; i++){
    CUDA_SAFE_CALL(hipMallocManaged(&x[i], n_mat * sizeof(int)));
  }

  fscanf(fp, "%*s"); // skip line

  for(int i=0; i < n_mat; i++)
  {
    for(int j=0; j < D; j++)
    {
        fscanf(fp, "%d %d %d", &val1, &val2, &val3);
        x[D*j][i] = val1;
        x[D*j+1][i] = val2;
        x[D*j+2][i] = val3;
    }
      fscanf(fp, "%*s");  // skip line
  }

  dim3 numBlocks(D*D);
  dim3 threadsPerBlock(n_mat);

  min_kernel<<<numBlocks, threadsPerBlock>>>(result, x, n_mat); //<<<number_of_blocks, block_size>>>

  hipDeviceSynchronize();
}

int main(int argc, char *argv[])
{

    char* filename = argv[1];
    int *result;
    int n_els = D*D;

    cudaReduction(result, filename)

    print_vector(result, n_els, D);

    return 0;
}

// int main(int argc, char *argv[])
// {
//     int **x;
//     int *y;
//     int n_els = D*D;
//     int n_mat;
//
//     FILE *fp;
//     int val1, val2, val3;
//
//     CUDA_SAFE_CALL(hipMallocManaged(&x, n_els * sizeof(int)));
//     CUDA_SAFE_CALL(hipMallocManaged(&y, n_els * sizeof(int)));
//
//     fp = fopen(argv[1], "r");
//     fscanf(fp, "%d", &n_mat);
//
//     for(int i=0; i < n_els; i++){
//       CUDA_SAFE_CALL(hipMallocManaged(&x[i], n_mat * sizeof(int)));
//     }
//
//     fscanf(fp, "%*s"); // skip line
//
//     for(int i=0; i < n_mat; i++)
//     {
//       for(int j=0; j < D; j++)
//       {
//           fscanf(fp, "%d %d %d", &val1, &val2, &val3);
//           x[D*j][i] = val1;
//           x[D*j+1][i] = val2;
//           x[D*j+2][i] = val3;
//       }
//         fscanf(fp, "%*s");  // skip line
//     }
//
//     dim3 numBlocks(D*D);
//     dim3 threadsPerBlock(n_mat);
//
// 	  min_kernel<<<numBlocks, threadsPerBlock>>>(y, x, n_mat); //<<<number_of_blocks, block_size>>>
//
//     hipDeviceSynchronize();
//
//     print_vector(y, n_els, D);
//
//     return 0;
// }
