/**
Ana Martinazzo (7209231)
Daniel
EP2 - redução em CUDA
**/

// ssh -R 52698:localhost:52698 anamartinazzo@shell.vision.ime.usp.br
// https://github.com/AJcodes/cuda_minmax/blob/master/cuda_minmax/kernel.cu

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define CUDA_SAFE_CALL(err) __cuda_safe_call(err, __FILE__, __LINE__)

#define D 3             // dimensão das matrizes (quadradas)
// #define inf 0x7f800000

int get_num_threads(int n);
void print_matrix(int** matrix, int n_rows, int n_cols);
void print_vector(int* vector, int n_els);

inline void
__cuda_safe_call (hipError_t err, const char *filename, const int line_number)
{
  if (err != hipSuccess)
    {
      printf ("CUDA error %i at %s:%i: %s\n",
          err, filename, line_number, hipGetErrorString (err));
      exit (-1);
    }
}


__global__ void min_kernel(int *result, int **input, int n_mat)
{
  //input (9x4)
  //tid=0 (prineira thread)
  //index=0 (primeiro elemente cada matriz)
  //input[index] = [4 1 2]
  //mintile[tid] = 4
  //
  // const int aaa = n_mat;
	__shared__ int mintile[4];
  for(int i=0; i<n_mat; i++)
    mintile[i] = 99;
	unsigned int tid = threadIdx.x;
	unsigned int index = blockIdx.x;
	mintile[tid] = input[index][2*tid];
  // if(mintile[tid] > 0)
	//   printf("i=%d, tid=%d, part_min=%d\n", index, tid, mintile[tid]);

  __syncthreads();

	// strided index and non-divergent branch
	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{

    int idx = 2*s*tid;
    // if(mintile[tid] > 0)
    //   printf("Dentro for: i=%d, tid=%d, s=%d, blockDim=%d\n",
    //        idx, tid, s, blockDim.x);
		if (idx < blockDim.x)
		{
      // if(mintile[tid] > 0)
      //   printf("primeiro if, i=%d, tid=%d, mintile[tid]=%d, input[tid + s]=%d\n", idx, tid, mintile[tid], input[index][2*tid + s]);
			if (mintile[tid + s] < mintile[tid])
      {
        // printf("Dentro if: i=%d, tid=%d, s=%d, mintile[tid]=%d, mintile[tid + s]=%d\n",
        //        idx, tid, s, mintile[tid], input[index][2*tid + s]);
        mintile[tid] = mintile[tid + s];
      }
		}
    if(mintile[tid] > 0)
    {
      printf("mintile:\n");
      for(int i=0; i<n_mat; i++)
        printf("%d ", mintile[i]);
      printf("\n");
    }
		__syncthreads();
	}
  // if(mintile[tid] > 0)
  //   printf("i=%d, tid=%d, part_min=%d\n", index, tid, mintile[tid]);

	if (tid == 0)
	{
		result[index] = mintile[0];
	}
}


int main(int argc, char *argv[])
{
    int **x;
    int *y;
    int n_els = D*D;
    int n_mat;

    FILE *fp;
    int val1, val2, val3;

    CUDA_SAFE_CALL(hipMallocManaged(&x, n_els * sizeof(int)));
    CUDA_SAFE_CALL(hipMallocManaged(&y, n_els * sizeof(int)));

    fp = fopen(argv[1], "r");
    fscanf(fp, "%d", &n_mat);

    // tratamento impar
    // if(n_mat%2 == 1)
    //   n_mat += 1;

    for(int i=0; i < n_els; i++){
      CUDA_SAFE_CALL(hipMallocManaged(&x[i], n_mat * sizeof(int)));
    }

    fscanf(fp, "%*s"); // pula linha

    for(int i=0; i < n_mat; i++)
    {
      for(int j=0; j < D; j++)
      {
          fscanf(fp, "%d %d %d", &val1, &val2, &val3);
          x[D*j][i] = val1;
          x[D*j+1][i] = val2;
          x[D*j+2][i] = val3;
      }
        fscanf(fp, "%*s"); // pula linha
    }

    //se impar, preenche com inf
    // for(int j=0; j < D; j++)
    // {
    //     x[D*j][n_mat] = INFINITY;
    //     x[D*j+1][n_mat] = INFINITY;
    //     x[D*j+2][n_mat] = INFINITY;
    // }

    print_matrix(x, n_els, n_mat);

    dim3 numBlocks(D*D);
    dim3 threadsPerBlock(n_mat);
    // <<<number_of_blocks, block_size>>>
	 min_kernel<<<numBlocks, threadsPerBlock>>>(y, x, n_mat);

    hipDeviceSynchronize();

    printf("y:\n");
    print_vector(y, n_els);

    return 0;

}


int get_num_threads(int n)
{
	// TODO: OTIMIZAR PRA WARP COM 32 THREADS ETC
	return ceil(n/2);
}


void print_matrix(int** matrix, int n_rows, int n_cols)
{
	printf("Printing matrix...\n");
	for(int i=0; i<n_rows; i++)
	{
		for(int j=0; j<n_cols; j++)
		{
			printf("%d ", matrix[i][j]);
		}
		printf("\n");
	}
}

void print_vector(int* vector, int n_els)
{
	printf("Printing vector...\n");
	for(int i=0; i<n_els; i++)
	{
		printf("%d ", vector[i]);
		if((i+1)%D == 0)
			printf("\n");
	}
	printf("\n");
}
