/**
Ana Martinazzo (7209231)
Daniel Sakuma (5619562)
EP2 - Redução em CUDA
**/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"
#include "reduction_seq.h"

#define MATRIX_ORDER 3 // ordem das matrizes (quadradas)

int main(int argc, char *argv[])
{
    char* filename = argv[1];

    int *resultCuda = cudaReduction(filename, MATRIX_ORDER);
    int *resultSeq = seqReduction(filename, MATRIX_ORDER);

    print_vector(resultCuda, MATRIX_ORDER);
    print_vector(resultSeq, MATRIX_ORDER);

    return 0;
}
