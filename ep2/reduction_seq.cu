#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_seq.h"


int* seqReduction(char filename[], int D)
{
  int* y;
  int n_els = D*D;

  /* allocate memory */
  y = malloc(n_els*sizeof(int*);

  /* initialize with zero */
  for(int i=0; i<n_els; i++){
    y[i] = 0;
  }

  return y;
}
