#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_seq.h"


int* reduction_seq(char filename[], int matrix_order)
{
  printf("Start reduction seq\n");
  int **x;
  int* y;
  int n_els = matrix_order*matrix_order;
  int n_mat = get_n_mat(filename);
  int val1, val2, val3;
  FILE *fp;

  /* allocate memory for y */
  y = (int*)malloc(n_els * sizeof(int));


  /* allocate memory for x */
  x = (int**)malloc(n_els*sizeof(int*));
  for(int i = 0; i<n_els; i++)
  {
    x[i] = (int*)malloc(n_mat * sizeof(int));
  }

  /* read matrix list */
  fp = fopen(filename, "r");
  fscanf(fp, "%*s");
  fscanf(fp, "%*s");
  for(int i=0; i < n_mat; i++)
  {
    for(int j=0; j < matrix_order; j++)
    {
        // printf("n_mat->%d, m_order->%d\n", i, j);

        fscanf(fp, "%d %d %d", &val1, &val2, &val3);
        // printf("%d %d %d\n", val1, val2, val3);
        // printf("place: %d %d %d %d\n", matrix_order*j, matrix_order*j+1, matrix_order*j+2, i);
        x[matrix_order*j][i] = val1;
        x[matrix_order*j+1][i] = val2;
        x[matrix_order*j+2][i] = val3;
        // printf("success\n");
    }
    fscanf(fp, "%*s");  // skip line
  }
  fclose(fp);

  /* initialize y with first matrix */
  for(int j=0; j < matrix_order; j++)
  {
      fscanf(fp, "%d %d %d", &val1, &val2, &val3);
      y[matrix_order*j] = val1;
      y[matrix_order*j+1] = val2;
      y[matrix_order*j+2] = val3;
      // printf("success\n");
  }
  fscanf(fp, "%*s");  // skip line

  /* sequential reduction */
  // for(int i=1; i < n_mat; i++)
  // {
  //   for(int j=0; j < matrix_order; j++)
  //   {
  //       fscanf(fp, "%d %d %d", &val1, &val2, &val3);
  //       x[matrix_order*j][i] = val1;
  //       x[matrix_order*j+1][i] = val2;
  //       x[matrix_order*j+2][i] = val3;
  //       // printf("success\n");
  //   }
  //   fscanf(fp, "%*s");  // skip line
  // }

  // print_matrix(x, n_els, n_mat);



  printf("Finish reduction seq\n");
  return y;
}
