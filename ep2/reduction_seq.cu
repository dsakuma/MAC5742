#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_seq.h"


int* reduction_seq(char filename[], int matrix_order)
{
  int **x;
  int* y;
  int n_els = matrix_order*matrix_order;
  int n_mat = get_n_mat(filename);
  int val1, val2, val3;
  FILE *fp;

  /* allocate memory for y */
  y = (int*)malloc(n_els * sizeof(int));


  /* allocate memory for x */
  x = (int**)malloc(n_els*sizeof(int*));
  for(int i = 0; i<n_els; i++)
  {
    x[i] = (int*)malloc(n_mat * sizeof(int));
  }

  /* read matrix list */
  fp = fopen(filename, "r");
  fscanf(fp, "%*s");
  fscanf(fp, "%*s");
  for(int i=0; i < n_mat; i++)
  {
    for(int j=0; j < matrix_order; j++)
    {
        fscanf(fp, "%d %d %d", &val1, &val2, &val3);
        x[matrix_order*j][i] = val1;
        x[matrix_order*j+1][i] = val2;
        x[matrix_order*j+2][i] = val3;
    }
    fscanf(fp, "%*s");  // skip line
  }
  fclose(fp);

  /* initialize y with first matrix */
  for(int j=0; j < matrix_order; j++)
  {
      y[matrix_order*j] = x[matrix_order*j][0];
      y[matrix_order*j+1] = x[matrix_order*j+1][0];
      y[matrix_order*j+2] = x[matrix_order*j+2][0];
  }

  /* sequential reduction */
  for(int i=1; i < n_mat; i++)
  {
    for(int j=0; j < matrix_order; j++)
    {
        if(x[matrix_order*j][i] < y[matrix_order*j])
          y[matrix_order*j] = x[matrix_order*j][i];
        if(x[matrix_order*j+1][i] < y[matrix_order*j+1])
          y[matrix_order*j+1] = x[matrix_order*j+1][i];
        if(x[matrix_order*j+2][i] < y[matrix_order*j+2])
          y[matrix_order*j+2] = x[matrix_order*j+2][i];
    }
  }

  return y;
}
