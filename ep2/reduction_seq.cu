#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_seq.h"


int* reduction_seq(char filename[], int matrix_order)
{
  printf("Start reduction seq\n");
  int **x;
  int* y;
  int n_els = matrix_order*matrix_order;
  int n_mat = get_n_mat(filename);
  int val1, val2, val3;
  FILE *fp;

  /* allocate memory for y */
  y = (int*)malloc(n_els * sizeof(int));


  /* allocate memory for x */
  x = (int**)malloc(n_els*sizeof(int));
  for(int i = 0; i<n_els; i++)
  {
    x[i] = (int*)malloc(n_mat * sizeof(int));
  }

  /* read matrix list */
  fp = fopen(filename, "r");
  fscanf(fp, "%*s");
  fscanf(fp, "%*s");
  // fscanf(fp, "%d %d %d", &val1, &val2, &val3);
  // printf("%d %d %d", val1, val2, val3);
  printf("n_mat->%d, m_order->%d\n" n_mat, matrix_order);

  // for(int i=0; i < n_mat; i++)
  // {
  //   for(int j=0; j < matrix_order; j++)
  //   {
  //       fscanf(fp, "%d %d %d", &val1, &val2, &val3);
  //       printf("%d %d %d", val1, val2, val3);
  //       x[matrix_order*j][i] = val1;
  //       x[matrix_order*j+1][i] = val2;
  //       x[matrix_order*j+2][i] = val3;
  //   }
  //     fscanf(fp, "%*s");  // skip line
  // }
  fclose(fp);

  // for(int i=0; i<matrix_order; i++){
  //   for(int j=0; j<n_els; j++){
  //       x[i][j] = 2;
  //     }
  // }

  // print_matrix(x, n_mat, n_els);


  // /* initialize with zero */
  // for(int i=0; i<n_els; i++){
  //   y[i] = 0;
  // }

  printf("Finish reduction seq\n");
  return y;
}
