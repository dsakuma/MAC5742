#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_seq.h"


int* reduction_seq(char filename[], int matrix_order)
{
  printf("Start reduction seq\n");
  int **x;
  int* y;
  int n_els = matrix_order*matrix_order;
  int n_mat = get_n_mat(filename);

  /* allocate memory for y */
  y = (int*)malloc(n_els * sizeof(int));


  /* allocate memory for x */
  x = (int**)malloc(n_mat*sizeof(int));
  for(int i = 0; i<n_mat; i++)
  {
    x[i] = (int*)malloc(n_els * sizeof(int));
  }

  /* initialize with zero */
  for(int i=0; i<matrix_order; i++){
    for(int j=0; j<n_els; j++){
        x[i][j] = 0;
      }
  }

  print_matrix(x, n_mat, n_els);


  // /* initialize with zero */
  // for(int i=0; i<n_els; i++){
  //   y[i] = 0;
  // }

  printf("Finish reduction seq\n");
  return y;
}
