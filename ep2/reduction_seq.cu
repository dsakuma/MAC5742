#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_seq.h"


int* seqReduction(char filename[], int D)
{
  int* y;

  /* allocate memory */
  y = malloc(n_rows*sizeof(int));

  /* initialize with zero */
  for(int i=0; i<D*D; i++){
    y[i] = 0;
  }

  return y;
}
