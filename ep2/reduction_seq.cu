#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_seq.h"


int* reduction_seq(char filename[], int D)
{
  int* y;
  int n_els = D*D;
  int n_mat = get_n_mat(filename);

  printf("n_mat: %d", n_mat);

  /* allocate memory */
  y = (int*)malloc(n_els * sizeof(int));

  /* initialize with zero */
  for(int i=0; i<n_els; i++){
    y[i] = 0;
  }

  return y;
}
