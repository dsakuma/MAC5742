#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"

#define THREADS_PER_BLOCK 2

__global__ void min_kernel(int *result, int **input, int n_mat)
{
	__shared__ int mintile[THREADS_PER_BLOCK];

	unsigned int tid = threadIdx.x;
	unsigned int index_x = blockIdx.x;
  unsigned int index_y = blockIdx.y;

  unsigned int start = (index_y*THREADS_PER_BLOCK)+tid;

  if(start >= n_mat)
    return;

	mintile[tid] = input[index_x][start];

  if(index_x ==0 && index_y==1)
  printf("index_x=%d (elem of mat),  index_y=%d (which partition), tid=%d (max 256), mintile[tid]=%d\n",
          index_x, index_y, tid, mintile[tid]);
  //x-> elemento
  //y-> particao
  //idx -> start
  //idx+s -> end

  __syncthreads();

	// strided index and non-divergent branch
	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
    int idx = 2*s*tid;
    if(index_x ==0 && index_y==1)
      printf("index_x=%d (elem of mat), index_y=%d (which partition), tid=%d (max 256), idx=%d, blockDim.x=%d, s=%d\n",
              index_x, index_y, tid, idx,blockDim.x, s);

    if(index_x ==0 && index_y==1)
      printf("+++ idx+s=%d, blockDim.x=%d, n_mat=%d", idx+s, blockDim.x, n_mat);

		if (idx+s < blockDim.x  && start+s < n_mat)
		{
      if(index_x ==0 && index_y==1)
        printf("index_x=%d (elem of mat), index_y=%d (which partition), tid=%d (max 256), idx=%d, blockDim.x=%d, s=%d, mintile[idx]=%d, mintile[idx+s]=%d\n",
                index_x, index_y, tid, idx,blockDim.x, s, mintile[idx], mintile[idx + s]);

			if (mintile[idx + s] < mintile[idx])
      {
        mintile[idx] = mintile[idx + s];
      }
		}
		__syncthreads();
	}

	if (tid == 0)
	{
    if(mintile[0] < result[index_x])
		  result[index_x] = mintile[0];
	}
}

int* reduction_cuda(const char filename[], int D)
{
  int **x;
  int *y;
  int n_els = D*D;
  int n_mat;

  FILE *fp;
  int val1, val2, val3;

  hipMallocManaged(&x, n_els * sizeof(int*));
  hipMallocManaged(&y, n_els * sizeof(int));

  fp = fopen(filename, "r");
  fscanf(fp, "%d", &n_mat);

  for(int i=0; i < n_els; i++){
    hipMallocManaged(&x[i], n_mat * sizeof(int));
  }

  fscanf(fp, "%*s"); // skip line

  for(int i=0; i < n_mat; i++)
  {
    for(int j=0; j < D; j++)
    {
        fscanf(fp, "%d %d %d", &val1, &val2, &val3);
        x[D*j][i] = val1;
        x[D*j+1][i] = val2;
        x[D*j+2][i] = val3;
    }
      fscanf(fp, "%*s");  // skip line
  }

  //teste
  y[0] = 999999999;
  y[1] = 999999999;
  y[2] = 999999999;
  y[3] = 999999999;
  y[4] = 999999999;
  y[5] = 999999999;
  y[6] = 999999999;
  y[7] = 999999999;
  y[8] = 999999999;
  // printf("nmat->%d\n", n_mat);
  // printf("threads->%d\n", THREADS_PER_BLOCK);
  // printf("ceil->%d\n", (int)ceil(n_mat/(float)THREADS_PER_BLOCK));
  dim3 numBlocks(n_els, (int)ceil(n_mat/(float)THREADS_PER_BLOCK));
  dim3 threadsPerBlock(THREADS_PER_BLOCK);

  min_kernel<<<numBlocks, threadsPerBlock>>>(y, x, n_mat); //<<<number_of_blocks, block_size>>>

  hipDeviceSynchronize();
  return y;
}
