#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"

#define CUDA_SAFE_CALL(err) __cuda_safe_call(err, __FILE__, __LINE__)

inline void
__cuda_safe_call (hipError_t err, const char *filename, const int line_number)
{
  if (err != hipSuccess)
    {
      printf ("CUDA error %i at %s:%i: %s\n",
          err, filename, line_number, hipGetErrorString (err));
      exit (-1);
    }
}

__global__ void min_kernel(int *result, int **input, int n_mat)
{
	__shared__ int mintile[9];

	unsigned int tid = threadIdx.x;
	unsigned int index = blockIdx.x;
	mintile[tid] = input[index][tid];

  printf("index=%d (bloco), tid=%d (n_mat), part_min=%d\n", index, tid, mintile[tid]);


  // if(mintile[tid] > 0)
  // {
  //   printf("mintile:\n");
  //   for(int i=0; i<n_mat; i++)
  //     printf("%d ", mintile[i]);
  //   printf("\n");
  // }

  __syncthreads();

	// strided index and non-divergent branch
	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
    int idx = 2*s*tid;
		if (idx < blockDim.x-1)
		{
			if (mintile[idx + s] < mintile[idx])
      {
        mintile[idx] = mintile[idx + s];
      }
		}
    if(tid == 0 && index ==1)
      printf("mintile[tid]=%d\n", mintile[tid]);
		__syncthreads();
    // if(mintile[tid] > 0)
    // {
    //   printf("mintile:\n");
    //   for(int i=0; i<n_mat; i++)
    //     printf("%d ", mintile[i]);
    //   printf("\n");
    // }
	}

	if (tid == 0)
	{
		result[index] = mintile[0];
	}
}

int* reduction_cuda(const char filename[], int D)
{
  int **x;
  int *y;
  int n_els = D*D;
  int n_mat;

  FILE *fp;
  int val1, val2, val3;

  CUDA_SAFE_CALL(hipMallocManaged(&x, n_els * sizeof(int*)));
  CUDA_SAFE_CALL(hipMallocManaged(&y, n_els * sizeof(int)));

  fp = fopen(filename, "r");
  fscanf(fp, "%d", &n_mat);

  for(int i=0; i < n_els; i++){
    CUDA_SAFE_CALL(hipMallocManaged(&x[i], n_mat * sizeof(int)));
  }

  fscanf(fp, "%*s"); // skip line

  for(int i=0; i < n_mat; i++)
  {
    for(int j=0; j < D; j++)
    {
        fscanf(fp, "%d %d %d", &val1, &val2, &val3);
        x[D*j][i] = val1;
        x[D*j+1][i] = val2;
        x[D*j+2][i] = val3;
    }
      fscanf(fp, "%*s");  // skip line
  }

  dim3 numBlocks(D*D);
  dim3 threadsPerBlock(n_mat);

  min_kernel<<<numBlocks, threadsPerBlock>>>(y, x, n_mat); //<<<number_of_blocks, block_size>>>

  hipDeviceSynchronize();
  return y;
}
