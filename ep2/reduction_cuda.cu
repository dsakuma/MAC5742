#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"

#define THREADS_PER_BLOCK 2

__global__ void min_kernel(int *result, int **input, int n_mat)
{
	__shared__ int mintile[THREADS_PER_BLOCK];

	unsigned int tid = threadIdx.x;
	unsigned int index_x = blockIdx.x;
  unsigned int index_y = blockIdx.y;

  unsigned int start = (index_y*THREADS_PER_BLOCK)+tid;
	mintile[tid] = input[index_x][start];

  printf("index_x=%d (elem of mat),  index_y=%d (which partition), tid=%d (max 256), mintile[tid]=%d\n",
          index_x, index_y, tid, mintile[tid]);
  //x-> elemento
  //y-> particao
  //idx -> start
  //idx+s -> end

  __syncthreads();

	// strided index and non-divergent branch
	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
    int idx = 2*s*tid;
    // if(tid == 2 && index ==1)
    //   printf("index=%d (bloco), tid=%d (n_mat), idx=%d, blockDim.x-1=%d\n", index, tid, idx,blockDim.x);
		if (idx+s < blockDim.x)
		{
      // if(tid == 2 && index ==1)
      //   printf("mintile[idx]=%d, mintile[idx+s]=%d\n", mintile[idx], mintile[idx + s]);

			if (mintile[idx + s] < mintile[idx])
      {
        mintile[idx] = mintile[idx + s];
      }
		}
		__syncthreads();
	}

	if (tid == 0)
	{
    if(mintile[0] < result[index_x])
		  result[index_x] = mintile[0];
	}
}

int* reduction_cuda(const char filename[], int D)
{
  int **x;
  int *y;
  int n_els = D*D;
  int n_mat;

  FILE *fp;
  int val1, val2, val3;

  hipMallocManaged(&x, n_els * sizeof(int*));
  hipMallocManaged(&y, n_els * sizeof(int));

  fp = fopen(filename, "r");
  fscanf(fp, "%d", &n_mat);

  for(int i=0; i < n_els; i++){
    hipMallocManaged(&x[i], n_mat * sizeof(int));
  }

  fscanf(fp, "%*s"); // skip line

  for(int i=0; i < n_mat; i++)
  {
    for(int j=0; j < D; j++)
    {
        fscanf(fp, "%d %d %d", &val1, &val2, &val3);
        x[D*j][i] = val1;
        x[D*j+1][i] = val2;
        x[D*j+2][i] = val3;
    }
      fscanf(fp, "%*s");  // skip line
  }

  //teste
  y[0] = 999999999;
  y[1] = 999999999;
  y[2] = 999999999;
  y[3] = 999999999;
  y[4] = 999999999;
  y[5] = 999999999;
  y[6] = 999999999;
  y[7] = 999999999;
  y[8] = 999999999;
  // printf("nmat->%d\n", n_mat);
  // printf("threads->%d\n", THREADS_PER_BLOCK);
  // printf("ceil->%d\n", (int)ceil(n_mat/(float)THREADS_PER_BLOCK));
  dim3 numBlocks(n_els, (int)ceil(n_mat/(float)THREADS_PER_BLOCK));
  dim3 threadsPerBlock(THREADS_PER_BLOCK);

  min_kernel<<<numBlocks, threadsPerBlock>>>(y, x, n_mat); //<<<number_of_blocks, block_size>>>

  hipDeviceSynchronize();
  return y;
}
