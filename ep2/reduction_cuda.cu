#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"

#define THREADS_PER_BLOCK 256

__global__ void min_kernel(int **result, int **input, int n_mat)
{
	__shared__ int mintile[THREADS_PER_BLOCK];

	unsigned int tid = threadIdx.x;
	unsigned int index_x = blockIdx.x;
  unsigned int index_y = blockIdx.y;

  unsigned int start = (index_y*THREADS_PER_BLOCK)+tid;

  if(start >= n_mat)
    return;

	mintile[tid] = input[index_x][start];

  // if(index_x ==0 )
  // printf("index_x=%d (elem of mat),  index_y=%d (partition), tid=%d (max 256), mintile[tid]=%d\n",
          // index_x, index_y, tid, mintile[tid]);
  //idx -> start
  //idx+s -> end

  __syncthreads();

	// strided index and non-divergent branch
	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
    int idx = 2*s*tid;
    // if(index_x ==0 && index_y==0)
      // printf("index_x=%d (elem of mat), index_y=%d (partition), tid=%d (max 256), idx=%d, blockDim.x=%d, s=%d\n",
              // index_x, index_y, tid, idx,blockDim.x, s);

    // if(index_x ==0 && index_y==0)
    //   printf("+++ idx+s=%d, blockDim.x=%d, n_mat=%d", idx+s, blockDim.x, n_mat);

		if (idx+s < blockDim.x  && start+s < n_mat)
		{
      // if(index_x ==0 && index_y==0)
      //   printf("index_x=%d (elem of mat), index_y=%d (partition), tid=%d (max 256), idx=%d, blockDim.x=%d, s=%d, mintile[idx]=%d, mintile[idx+s]=%d\n",
      //           index_x, index_y, tid, idx,blockDim.x, s, mintile[idx], mintile[idx + s]);
      // myLock.lock(index_x);
			if (mintile[idx + s] < mintile[idx])
      {
        mintile[idx] = mintile[idx + s];
      }
      // myLock.unlock(index_x);
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		 result[index_x][index_y] = mintile[0];
	}
}

int* reduction_cuda(const char filename[], int D)
{
  int **x;
  int **y;
	int *res;
  int n_els = D*D;
  int n_mat;
	int n_partitions;
  // Lock myLock;

  FILE *fp;
  int val1, val2, val3;

  hipMallocManaged(&x, n_els * sizeof(int*));
  hipMallocManaged(&y, n_els * sizeof(int*));

  fp = fopen(filename, "r");
  fscanf(fp, "%d", &n_mat);

  for(int i=0; i < n_els; i++){
    hipMallocManaged(&x[i], n_mat * sizeof(int));
  }

	for(int i=0; i < n_els; i++){
    hipMallocManaged(&y[i], n_mat * sizeof(int));
  }

  fscanf(fp, "%*s"); // skip line

  for(int i=0; i < n_mat; i++)
  {
    for(int j=0; j < D; j++)
    {
        fscanf(fp, "%d %d %d", &val1, &val2, &val3);
        x[D*j][i] = val1;
        x[D*j+1][i] = val2;
        x[D*j+2][i] = val3;
    }
      fscanf(fp, "%*s");  // skip line
  }

  //teste
  y[0][0] = 999999999;
  y[1][0] = 999999999;
  y[2][0] = 999999999;
  y[3][0] = 999999999;
  y[4][0] = 999999999;
  y[5][0] = 999999999;
  y[6][0] = 999999999;
  y[7][0] = 999999999;
  y[8][0] = 999999999;
  // printf("nmat->%d\n", n_mat);
  // printf("threads->%d\n", THREADS_PER_BLOCK);
  // printf("ceil->%d\n", (int)ceil(n_mat/(float)THREADS_PER_BLOCK));

	do{
		n_partitions = (int)ceil(n_mat/(float)THREADS_PER_BLOCK);
		dim3 numBlocks(n_els, n_partitions);
		dim3 threadsPerBlock(THREADS_PER_BLOCK);
		// printf("Chamando o kernel\n");
		min_kernel<<<numBlocks, threadsPerBlock>>>(y, x, n_mat); //<<<number_of_blocks, block_size>>>
		hipDeviceSynchronize();
		n_mat = n_partitions;

		for(int i=0; i < n_els; i++)
		{
			for(int j=0; j < n_mat; j++)
			{
					x[i][j] = y[i][j];
			}
		}

	}while(n_partitions > 1);


  // return y;

	res = (int*) calloc(n_els, sizeof(int));
	for(int i=0; i < n_els; i++)
			res[i] = y[i][0];

	return res;
}
