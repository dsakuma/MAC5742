#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"

#define BLOCKSIZE 32
#define BLOCKSIZE_Z 1024    // avoid bus error when number of threads > 1024

#define CUDA_SAFE_CALL(err) __cuda_safe_call(err, __FILE__, __LINE__)

inline void
__cuda_safe_call (hipError_t err, const char *filename, const int line_number)
{
  if (err != hipSuccess)
    {
      printf ("CUDA error %i at %s:%i: %s\n",
          err, filename, line_number, hipGetErrorString (err));
      exit (-1);
    }
}

__global__ void partial_min(int **result, int **input)
{
	__shared__ int mintile[BLOCKSIZE];

	unsigned int tid = threadIdx.x;
	unsigned int index_x = blockIdx.x;
  unsigned int index_y = blockIdx.y;
  unsigned int index_z = blockIdx.z;
	mintile[tid] = input[index_x][BLOCKSIZE_Z*index_z + BLOCKSIZE*index_y + tid];

	if(tid==0)
		//printf("index=%d (bloco), tid=%d (n_mat), mintile=%d, blockDim.x=%d\n", index_x, tid, mintile[tid], blockDim.x);

  	__syncthreads();

	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
        int idx = 2*s*tid;
		if (idx+s < blockDim.x)
		{
			if (mintile[idx + s] < mintile[idx])
            {
                mintile[idx] = mintile[idx + s];
            }
		}
		__syncthreads();
	}

	if (tid == 0)
	{	
		//printf("result[%d][%d]=%d\n", index_x, (index_z+1)*index_y, mintile[0]);
		result[index_x][index_z*BLOCKSIZE + index_y] = mintile[0];
	}
}


__global__ void final_min(int *result, int **input)
{
    __shared__ int mintile[BLOCKSIZE_Z];

    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x;
    mintile[tid] = input[index][tid];

    __syncthreads();

    // strided index and non-divergent branch
    for (unsigned int s = 1; s < blockDim.x; s *= 2)
    {
        int idx = 2*s*tid;
        if (idx+s < blockDim.x)
        {
            if (mintile[idx + s] < mintile[idx])
            {
                mintile[idx] = mintile[idx + s];
            }
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        result[index] = mintile[0];
    }
}


int* reduction_cuda(const char filename[], int D)
{
  int **x;
  int **y;
  int *res;
  int n_els = D*D;
  int n_mat, blocks_y, blocks_z, total_blocks;

  FILE *fp;
  int val1, val2, val3;

  CUDA_SAFE_CALL(hipMallocManaged(&x, n_els * sizeof(int*)));
  CUDA_SAFE_CALL(hipMallocManaged(&y, n_els * sizeof(int*)));
  CUDA_SAFE_CALL(hipMallocManaged(&res, n_els * sizeof(int)));

  fp = fopen(filename, "r");
  fscanf(fp, "%d", &n_mat);
  blocks_z = ceil(n_mat/(float) BLOCKSIZE_Z);
  blocks_y = ceil(n_mat/(float) (blocks_z*BLOCKSIZE));
  total_blocks = blocks_y*blocks_z;

  printf("blocks_z: %d, blocks_y: %d, total: %d\n", blocks_z, blocks_y, total_blocks);  

  for(int i=0; i < n_els; i++){
    CUDA_SAFE_CALL(hipMallocManaged(&x[i], n_mat * sizeof(int)));
    CUDA_SAFE_CALL(hipMallocManaged(&y[i], total_blocks * sizeof(int)));
  }

  fscanf(fp, "%*s"); // skip line

  for(int i=0; i < n_mat; i++)
  {
    for(int j=0; j < D; j++)
    {
        fscanf(fp, "%d %d %d", &val1, &val2, &val3);
        x[D*j][i] = val1;
        x[D*j+1][i] = val2;
        x[D*j+2][i] = val3;
    }
      fscanf(fp, "%*s");  // skip line
  }

  dim3 numBlocks(n_els, blocks_y, blocks_z);
  dim3 threadsPerBlock(BLOCKSIZE);

  partial_min<<<numBlocks, threadsPerBlock>>>(y, x); //<<<number_of_blocks, block_size>>>
  final_min<<<n_els, total_blocks>>>(res, y);

  hipDeviceSynchronize();

  //print_matrix(y, n_els, total_blocks);
  
  return res;
}
