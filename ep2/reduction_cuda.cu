#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


int main(int argc, char **argv)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    return 0;
}
