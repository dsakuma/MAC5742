#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"

#define CUDA_SAFE_CALL(err) __cuda_safe_call(err, __FILE__, __LINE__)

inline void
__cuda_safe_call (hipError_t err, const char *filename, const int line_number)
{
  if (err != hipSuccess)
    {
      printf ("CUDA error %i at %s:%i: %s\n",
          err, filename, line_number, hipGetErrorString (err));
      exit (-1);
    }
}

__global__ void min_kernel(int *result, int **input, int n_mat)
{
	__shared__ int mintile[1023];

	unsigned int tid = threadIdx.x;
	unsigned int index = blockIdx.x;
	mintile[tid] = input[index][tid];

  // printf("index=%d (bloco), tid=%d (n_mat), part_min=%d, blockDim.x-1=%d\n", index, tid, mintile[tid], blockDim.x);

  __syncthreads();

	// strided index and non-divergent branch
	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
    int idx = 2*s*tid;
    // if(tid == 2 && index ==1)
    //   printf("index=%d (bloco), tid=%d (n_mat), idx=%d, blockDim.x-1=%d\n", index, tid, idx,blockDim.x);
		if (idx+s < blockDim.x)
		{
      // if(tid == 2 && index ==1)
      //   printf("mintile[idx]=%d, mintile[idx+s]=%d\n", mintile[idx], mintile[idx + s]);

			if (mintile[idx + s] < mintile[idx])
      {
        mintile[idx] = mintile[idx + s];
      }
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		result[index] = mintile[0];
	}
}

int* reduction_cuda(const char filename[], int D)
{
  int **x;
  int *y;
  int n_els = D*D;
  int n_mat;

  FILE *fp;
  int val1, val2, val3;

  CUDA_SAFE_CALL(hipMallocManaged(&x, n_els * sizeof(int*)));
  CUDA_SAFE_CALL(hipMallocManaged(&y, n_els * sizeof(int)));

  fp = fopen(filename, "r");
  fscanf(fp, "%d", &n_mat);

  for(int i=0; i < n_els; i++){
    CUDA_SAFE_CALL(hipMallocManaged(&x[i], n_mat * sizeof(int)));
  }

  fscanf(fp, "%*s"); // skip line

  for(int i=0; i < n_mat; i++)
  {
    for(int j=0; j < D; j++)
    {
        fscanf(fp, "%d %d %d", &val1, &val2, &val3);
        x[D*j][i] = val1;
        x[D*j+1][i] = val2;
        x[D*j+2][i] = val3;
    }
      fscanf(fp, "%*s");  // skip line
  }

  dim3 numBlocks(D*D);
  dim3 threadsPerBlock(n_mat);

  min_kernel<<<numBlocks, threadsPerBlock>>>(y, x, n_mat); //<<<number_of_blocks, block_size>>>

  hipDeviceSynchronize();
  return y;
}
