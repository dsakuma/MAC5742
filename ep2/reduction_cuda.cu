#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"

#define THREADS_PER_BLOCK 256

__global__ void min_kernel(int **input, int n_mat)
{
	__shared__ int mintile[THREADS_PER_BLOCK];

	unsigned int tid = threadIdx.x;
	unsigned int index_x = blockIdx.x;
	unsigned int index_y = blockIdx.y;
	unsigned int start = (index_y*THREADS_PER_BLOCK)+tid;

	if(start >= n_mat)
		return;

	mintile[tid] = input[index_x][start];
	__syncthreads();

	if(tid%2 != 0)
		return;

	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
		int idx = tid;
		if (idx+s < blockDim.x  && start+s < n_mat)
			mintile[idx] = (abs(mintile[idx]+mintile[idx+s]) - abs(mintile[idx]-mintile[idx+s]))/2;
		__syncthreads();
	}

	if (tid == 0)
		input[index_x][index_y] = mintile[0];
}

int* reduction_cuda(const char filename[], int D)
{
	int **x;
	int *res;
	int n_els = D*D;
	int n_mat;
	int n_partitions;

	FILE *fp;
	int val1, val2, val3;

	fp = fopen(filename, "r");
	fscanf(fp, "%d", &n_mat);

	/* allocate memory */
	hipMallocManaged(&x, n_els * sizeof(int*));
	for(int i=0; i < n_els; i++)
		hipMallocManaged(&x[i], n_mat * sizeof(int));

	/* read matrix list */
	fscanf(fp, "%*s"); // skip line
	for(int i=0; i < n_mat; i++)
	{
		for(int j=0; j < D; j++)
		{
			fscanf(fp, "%d %d %d", &val1, &val2, &val3);
			x[D*j][i] = val1;
			x[D*j+1][i] = val2;
			x[D*j+2][i] = val3;
		}
		fscanf(fp, "%*s");  // skip line
	}

	/* cuda reduction */
	do {
		n_partitions = (int)ceil(n_mat/(float)THREADS_PER_BLOCK);
		dim3 numBlocks(n_els, n_partitions);
		dim3 threadsPerBlock(THREADS_PER_BLOCK);
		min_kernel<<<numBlocks, threadsPerBlock>>>(x, n_mat); //<<<number_of_blocks, block_size>>>
		n_mat = n_partitions;
	} while (n_partitions > 1);

	hipDeviceSynchronize();

	/* get reduced matrix */
	res = (int*) calloc(n_els, sizeof(int));
	for(int i=0; i < n_els; i++)
		res[i] = x[i][0];

	hipFree(x);

	return res;
}
