#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"

#define THREADS_PER_BLOCK 256

__global__ void min_kernel(int *result, int **input, int n_mat)
{
	__shared__ int mintile[9];

	unsigned int tid = threadIdx.x;
	unsigned int index = blockIdx.x;
	mintile[tid] = input[index][tid];

  // printf("index=%d (bloco), tid=%d (n_mat), part_min=%d, blockDim.x-1=%d\n", index, tid, mintile[tid], blockDim.x);

  __syncthreads();

	// strided index and non-divergent branch
	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
    int idx = 2*s*tid;
    // if(tid == 2 && index ==1)
    //   printf("index=%d (bloco), tid=%d (n_mat), idx=%d, blockDim.x-1=%d\n", index, tid, idx,blockDim.x);
		if (idx+s < blockDim.x)
		{
      // if(tid == 2 && index ==1)
      //   printf("mintile[idx]=%d, mintile[idx+s]=%d\n", mintile[idx], mintile[idx + s]);

			if (mintile[idx + s] < mintile[idx])
      {
        mintile[idx] = mintile[idx + s];
      }
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		result[index] = mintile[0];
	}
}

int* reduction_cuda(const char filename[], int D)
{
  int **x;
  int *y;
  int n_els = D*D;
  int n_mat;

  FILE *fp;
  int val1, val2, val3;

  hipMallocManaged(&x, n_els * sizeof(int*));
  hipMallocManaged(&y, n_els * sizeof(int));

  fp = fopen(filename, "r");
  fscanf(fp, "%d", &n_mat);

  for(int i=0; i < n_els; i++){
    hipMallocManaged(&x[i], n_mat * sizeof(int));
  }

  fscanf(fp, "%*s"); // skip line

  for(int i=0; i < n_mat; i++)
  {
    for(int j=0; j < D; j++)
    {
        fscanf(fp, "%d %d %d", &val1, &val2, &val3);
        x[D*j][i] = val1;
        x[D*j+1][i] = val2;
        x[D*j+2][i] = val3;
    }
      fscanf(fp, "%*s");  // skip line
  }

  printf("nmat->%d\n", n_mat);
  printf("threads->%d\n", THREADS_PER_BLOCK);
  printf("ceil->%d\n", ceil(n_mat/(float)THREADS_PER_BLOCK));
  dim3 numBlocks(n_els, ceil(n_mat/THREADS_PER_BLOCK));
  dim3 threadsPerBlock(THREADS_PER_BLOCK);

  min_kernel<<<numBlocks, threadsPerBlock>>>(y, x, n_mat); //<<<number_of_blocks, block_size>>>

  hipDeviceSynchronize();
  return y;
}
