#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include "functions.h"
#include "reduction_cuda.h"

#define BLOCKSIZE 256

#define CUDA_SAFE_CALL(err) __cuda_safe_call(err, __FILE__, __LINE__)

inline void
__cuda_safe_call (hipError_t err, const char *filename, const int line_number)
{
  if (err != hipSuccess)
    {
      printf ("CUDA error %i at %s:%i: %s\n",
          err, filename, line_number, hipGetErrorString (err));
      exit (-1);
    }
}

__global__ void min_reduction(int **result, int **input, int n_mat)
{
	__shared__ int mintile[BLOCKSIZE];

    unsigned int n_elements = n_mat;
    unsigned int tid = threadIdx.x;
    unsigned int index_x = blockIdx.x;
    unsigned int index_y = blockIdx.y * blockDim.y;
    unsigned int last_pos;
    
    // for(int n_elements = n_mat; n_elements > BLOCKSIZE; n_elements /= ceil(n_elements/(float)BLOCKSIZE))
    while(n_elements > 1)
    {
        for(int i = index_y + tid; i < n_elements; i += BLOCKSIZE)
        {
        	mintile[tid] = input[index_x][i];

            __syncthreads();

        	// if(tid==0)
        	// 	printf("index_x=%d, index_y=%d, tid=%d, mintile=%d, blockDim.x=%d, blockDim.y=%d, gridDim.x=%d, gridDim.y=%d\n",
         //            index_x, index_y, tid, mintile[tid], blockDim.x, blockDim.y, gridDim.x, gridDim.y);

            // if(blockIdx.y == gridDim.y-1) {
            //     last_pos = n_elements < BLOCKSIZE ? n_elements : BLOCKSIZE; // ultimo bloco em y
            // } else {
            //     last_pos = stride*BLOCKSIZE;
            // }

            last_pos = n_elements < BLOCKSIZE ? n_elements : BLOCKSIZE;

        	for (unsigned int s = 1; s < last_pos; s *= 2)
        	{
                int idx = 2*s*tid;
        		if (idx+s < last_pos)
        		{
        			if (mintile[idx+s] < mintile[idx])
                        mintile[idx] = mintile[idx+s];
        		}
        		__syncthreads();
        	}

        	if (tid == 0)
        	{	
        		// printf("last_pos=%d, i=%d, mintile[0]=%d\n", last_pos, i, mintile[0]);
        		input[index_x][index_y] = mintile[0];
        	}
        }
        n_elements = ceil(n_elements/(float)BLOCKSIZE);
    }
}


int* reduction_cuda(const char filename[], int D)
{
    int **x, **y, *res;
    int n_mat, ysize, numSMs;
    int n_els = D*D;

    FILE *fp;
    int val1, val2, val3;

    CUDA_SAFE_CALL(hipMallocManaged(&x, n_els * sizeof(int*)));
    CUDA_SAFE_CALL(hipMallocManaged(&y, n_els * sizeof(int*)));

    fp = fopen(filename, "r");
    fscanf(fp, "%d", &n_mat);
    ysize = ceil(n_mat/(float)BLOCKSIZE);

    for(int i=0; i < n_els; i++)
    {
        CUDA_SAFE_CALL(hipMallocManaged(&x[i], n_mat * sizeof(int)));
        CUDA_SAFE_CALL(hipMallocManaged(&y[i], ysize * sizeof(int)));
    }

    fscanf(fp, "%*s"); // skip line

    for(int i=0; i < n_mat; i++)
    {
        for(int j=0; j < D; j++)
        {
            fscanf(fp, "%d %d %d", &val1, &val2, &val3);
            x[D*j][i] = val1;
            x[D*j+1][i] = val2;
            x[D*j+2][i] = val3;
        }
        fscanf(fp, "%*s");  // skip line
    }

    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
    dim3 numBlocks(n_els, 32*numSMs);
    dim3 threadsPerBlock(BLOCKSIZE);

    printf("n_els=%d, numSMs=%d, BLOCKSIZE=%d\n", n_els, numSMs, BLOCKSIZE);

    // a funcao sobrescreve x e nao usa y
    min_reduction<<<numBlocks, threadsPerBlock>>>(y, x, n_mat); //<<<number_of_blocks, block_size>>>
    // min_reduction<<<2,2>>>(y, x, n_mat);

    hipDeviceSynchronize();

    res = (int*) calloc(n_els, sizeof(int));
    for(int i=0; i < n_els; i++)
        res[i] = x[i][0];

    return res;
}
